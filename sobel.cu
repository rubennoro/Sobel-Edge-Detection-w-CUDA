
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define THRESHOLD 128
#define BLOCKS 16
#define THREADS 32

__global__ void sobel_gpu(float* intensity, float* output, int height, int width){
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    float dx, dy;

    if(row > 0 && col > 0 && col < height-1 && row < width-1){
    	
	dx = (intensity[(row-1)*width+(col+1)]+2*intensity[(row)*width+(col+1)]+intensity[(row+1)*width+(col+1)]) - (intensity[(row-1)*width+(col-1)]+2*intensity[(row)*width+(col-1)]+intensity[(row+1)*width+(col-1)]);
	
	dy = (intensity[(row-1)*width+(col-1)]+2*intensity[(row-1)*width+col]+intensity[(row-1)*width+(col+1)]) - (intensity[(row+1)*width+(col-1)]+2*intensity[(row+1)*width+(col)]+intensity[(row+1)*width+(col+1)]);


	//dx = (intensity[i-1][j+1]+2*intensity[i][j+1]+intensity[i+1][j+1])-(intensity[i-1][j-1]+2*intensity[i][j-1]+intensity[i+1][j-1]);
        //dy = (intensity[i-1][j-1]+2*intensity[i-1][j]+intensity[i-1][j-1])-(intensity[i+1][j-1]+2*intensity[i+1][j]+intensity[i+1][j+1]);
        //print("dx: %f, dy: %f", dx, dy);                                                                                                                                                                                                                                          
        output[row*width+col] = (float)sqrt((dx*dx)+(dy*dy));

         
    }    
}

int main(){

    FILE *file_in = fopen("bmp_file_exp.bmp", "r");
    FILE *file_sobel = fopen("bmp_out.bmp", "w+");

    if(file_in == NULL){
        printf("File is not found");
    }
    int i, j;
    unsigned char bytes[54];
    for(i = 0; i < 54; i++){
    	  bytes[i] = getc(file_in);
    }
    fwrite(bytes, sizeof(unsigned char), 54, file_in);
    fwrite(bytes, sizeof(unsigned char), 54, file_sobel);
    
    int height = *(int *)&bytes[18];
    int width = *(int *)&bytes[22];
    
    int size = width * height;
    printf("WidthxHeight: %d\n", size);
    float* intensity;
    float* sobel;
    hipHostMalloc((void **) &intensity, sizeof(float) * size, hipHostMallocDefault);
    hipHostMalloc((void **) &sobel, sizeof(float) * size, hipHostMallocDefault);
 
    //float* intensity = malloc(size * sizeof(float));
    //float* sobel = malloc(size * sizeof(float));
    unsigned char r, g, b;
    
    //Calculation for Intensity of operation file
    for(i = 0; i < size; i++){
    	  b = getc(file_in);
	  g = getc(file_in);
	  r = getc(file_in);
    	  intensity[i] = 0.299 * r + 0.587 * g + 0.114 * b;
	  sobel[i] = 0;
    }
    //Tracking time it took
    float gpu_time_ms;

    float* sobel_cu;
    float* intensity_cu;
    
    hipMalloc((void **) &sobel_cu, size * sizeof(float));
    hipMalloc((void **) &intensity_cu, size * sizeof(float));
    
    hipMemcpy(sobel_cu, sobel, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(intensity_cu, intensity, sizeof(float) * size, hipMemcpyHostToDevice);
    
    unsigned int threads_per = THREADS;
    
    //MATH TO FULLY COVER THE MATRIX
    unsigned int blocks_per_x = (width+THREADS)-1/(THREADS);
    unsigned int blocks_per_y = (height+THREADS)-1/(THREADS);
    
    //DEFINE THESE VALUES
    dim3 threadsPerBlock(threads_per, threads_per);
    dim3 numBlocks(blocks_per_x, blocks_per_y);
    
    printf("here\n");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    
    hipEventRecord(start, 0);

    sobel_gpu<<<numBlocks, threadsPerBlock>>>(intensity_cu, sobel_cu, height, width); 

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipMemcpy(sobel, sobel_cu, sizeof(float)*size, hipMemcpyDeviceToHost);
    
    hipEventElapsedTime(&gpu_time_ms, start, stop);
    printf("Time: %f\n", gpu_time_ms);
    float final_val;
    for(i = 0; i < size; i++){
    	  
    	  final_val = sobel[i] > THRESHOLD? 255 : 0;
	  putc(final_val, file_sobel);
	  putc(final_val, file_sobel);
	  putc(final_val, file_sobel);
    }

    hipHostFree(intensity);
    hipHostFree(sobel);
    hipFree(sobel_cu);
    hipFree(intensity_cu);
    fclose(file_sobel);
    fclose(file_in);
    return 0;    

}
